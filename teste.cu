
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include<cstring>
 
    
#define N 10;
    
__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
}
        }

int main( int argc, char **argv ){
    float *a, *b, *out; 

    // Allocate memory
    int vet1[10];
    int vet2[10];
    int res[10];
    a   = (float*)malloc(sizeof(float) * 10);
    b   = (float*)malloc(sizeof(float) * 10);
    out = (float*)malloc(sizeof(float) * 10);

    
    // Initialize array
    for(int i = 0; i < 10; i++){
        vet1[i] = 1.0f; vet2[i] = 2.0f;
    }
    
    hipMalloc( (void**) &a, 10 * sizeof(float));
    hipMalloc( (void**) &b, 10 * sizeof(float));
    hipMalloc( (void**) &out, 10 * sizeof(float));
 
    hipMemcpy( a,vet1, 10 * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( b,vet2, 10 * sizeof(int), hipMemcpyHostToDevice );
        
    vector_add<<<1, 10>>>( out, a, b, 10 );
    
     hipMemcpy( res, out, 10 * sizeof(int), hipMemcpyDeviceToHost );
     
    for(int i = 0; i < 10; i++ ){
        printf( "%d ", res[i] );
    }
 
    puts("");
 
    return 0;

}

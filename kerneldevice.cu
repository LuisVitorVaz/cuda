
#include <hip/hip_runtime.h>
#include <stdio.h>

// Função definida pelo programador que será chamada no kernel
__device__ int minhaFuncaoGPU(int a, int b) {
    return a + b;
}

// Kernel CUDA: Chama a função na GPU
__global__ void meuKernel(int *resultado) {
    int threadId = threadIdx.x;
    int blocoId = blockIdx.x;

    // Chamando a função na GPU
    resultado[threadId] = minhaFuncaoGPU(threadId, blocoId);
}

int main() {
    const int tamanho = 10;
    int resultado[tamanho];

    int *d_resultado;  // Ponteiro para o resultado no device

    // Aloca memória no device
    hipMalloc((void**)&d_resultado, tamanho * sizeof(int));

    // Configuração de lançamento do kernel
    dim3 blocos(1);
    dim3 threads(tamanho);

    // Chama o kernel no device
    meuKernel<<<blocos, threads>>>(d_resultado);

    // Copia os resultados do device para o host
    hipMemcpy(resultado, d_resultado, tamanho * sizeof(int), hipMemcpyDeviceToHost);

    // Imprime os resultados
    printf("Resultados:\n");
    for (int i = 0; i < tamanho; ++i) {
        printf("%d\n", resultado[i]);
    }

    // Libera a memória alocada no device
    hipFree(d_resultado);

    return 0;
}


#include <hip/hip_runtime.h>
#include <iostream>
 const int N = 10;
 // Tamanho do vetor 
// Kernel para somar os elementos do vetor (uma única thread) 
__global__ void somaVetor(int *vetor, int *resultado) { 
if (threadIdx.x == 0) {
 for (int i = 0; i < N; ++i) { resultado[0] += vetor[i]; } 
    
} 
    
}

 int main() {
     
 const int tamanhoVetor = N; 
 int vetor[tamanhoVetor] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
 int resultado[1] = {0}; 
// A soma será armazenada aqui
 int *d_vetor, *d_resultado;
 // Aloca memória na GPU
 hipMalloc((void**)&d_vetor, sizeof(int) * tamanhoVetor);
 hipMalloc((void**)&d_resultado, sizeof(int)); 
// Copia dados do host para a GPU
 hipMemcpy(d_vetor, vetor, sizeof(int) * tamanhoVetor, hipMemcpyHostToDevice); 
// Chama o kernel com um bloco e uma única thread 
somaVetor<<<1, 1>>>(d_vetor, d_resultado); 
// Copia o resultado de volta para o host 
hipMemcpy(resultado, d_resultado, sizeof(int), hipMemcpyDeviceToHost);
 std::cout << "A soma dos elementos do vetor é: " << resultado[0] << std::endl;
 // Libera memória cudaFree(d_vetor); cudaFree(d_resultado); return 0;
 }

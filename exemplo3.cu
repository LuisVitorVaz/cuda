
#include <hip/hip_runtime.h>
#include <iostream>

const int N = 1024;  // Tamanho do vetor

// Kernel para somar os elementos do vetor

__global__ void somaVetor(int *vetor, int *resultado) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Cada thread soma um par de elementos adjacentes
    if (idx < N) {
        resultado[idx] = vetor[idx];
    }
}

int main() {
    const int tamanhoVetor = N; //e 10
    int vetor[tamanhoVetor] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    int resultado[tamanhoVetor];


    int *d_vetor, *d_resultado;

    // Aloca memória na GPU
    hipMalloc((void**)&d_vetor, sizeof(int) * tamanhoVetor);
    hipMalloc((void**)&d_resultado, sizeof(int) * tamanhoVetor);

    // Copia dados do host para a GPU
    hipMemcpy(d_vetor, vetor, sizeof(int) * tamanhoVetor, hipMemcpyHostToDevice);

    // Chama o kernel com um bloco e threads por bloco suficientes
    somaVetor<<<1, tamanhoVetor>>>(d_vetor, d_resultado);//<<<1,1>>> primeiro o bloco depois a thread

    // Copia o resultado de volta para o host
    hipMemcpy(resultado, d_resultado, sizeof(int) * tamanhoVetor, hipMemcpyDeviceToHost);

    // Soma final no host
    int resultadoFinal = 0;
    for (int i = 0; i < tamanhoVetor; ++i) {
        resultadoFinal += resultado[i];
    }

    std::cout << "A soma dos elementos nesse  vetor é: " << resultadoFinal << std::endl;
    // Libera memória
    hipFree(d_vetor);
    hipFree(d_resultado);

    return 0;
}

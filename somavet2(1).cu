
#include <hip/hip_runtime.h>
#include <stdio.h>

// Device code
__global__ void VecAdd(float *A, float* B, float* C, int n){
     int i =blockIdx.x; 
     if (i < n)
         C[i] = A[i] + B[i];
}

// Host code
int main()	{
int n =700;
size_t size = n * sizeof(float); 
float *d_A, *d_B, *d_C;
float h_A[n] ;
float h_B[n] ;
float h_C[n] ;

hipMalloc((void**)&d_A, size);
hipMalloc((void**)&d_B, size);
hipMalloc((void**)&d_C, size);
/*
float h_A[] = {1,2,3,4,5};
float h_B[] = {10,20,30,40,50};
float h_C[] = {0,0,0,0,0};
*/
for (int i = 0; i<n; i++) {
    h_A[i] = 1;
    h_B[i] = 1;
    h_C[i] = 0;
}

hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

int nThreadsPerBlock	= 1;
int nBlocks	= n ; //n /nThreadsPerBlock;

VecAdd<<<nBlocks, nThreadsPerBlock>>>(d_A, d_B, d_C, n);

hipMemcpy(h_C, d_C, size,hipMemcpyDeviceToHost);	


printf ("Vetor resultado: \n") ;

for (int i = 0; i<n-1; i++) {
    printf ("%.0f ", h_C[i]) ;
}

h_C[n] = 555 ;

printf ("Valor ultimo elemento: %.0f\n", h_C[n]) ;

hipFree(d_B);
hipFree(d_C);
}

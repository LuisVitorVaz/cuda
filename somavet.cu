
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include<cstring>
 
__global__ void vecAdd( int *v1, int *v2, int *res ){
    int i = threadIdx.x;
    res[i] = v1[i] * v2[i];
}
 
// teste simples
 
int main( int argc, char **argv ){
    int array[]={2,4,6,8,10, 12,14,16,18,20};
    int array2[]={1,1,1,1,1, 1,1,1,1,1};
 
    int *cudaArr1, *cudaArr2, *resCuda;
 
    int res[10];
 
    //cudaMemCpy(  );
 
    hipMalloc( (void**) &cudaArr1, 10 * sizeof(int));
    hipMalloc( (void**) &cudaArr2, 10 * sizeof(int));
    hipMalloc( (void**) &resCuda, 10 * sizeof(int));
 
    hipMemcpy( cudaArr1, array, 10 * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( cudaArr2, array2, 10 * sizeof(int), hipMemcpyHostToDevice );
 
    vecAdd<<<1, 10>>>( cudaArr1, cudaArr2, resCuda );
 
    //cudaMemcpy( res, cudaArr1, 5* sizeof(int), cudaMemcpyDeviceToHost );
    hipMemcpy( res, resCuda, 10* sizeof(int), hipMemcpyDeviceToHost );
 
    for( int i = 0; i < 10; i++ ){
        printf( "%d ", res[i] );
    }
 
    puts("");
 
    return 0;
}
